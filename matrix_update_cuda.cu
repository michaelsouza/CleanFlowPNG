#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

// CUDA kernel for updating the matrix
__global__ void updateMatrixKernel(int* d_matrix, int* d_new_matrix, int m, int n, bool* d_updated) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        int current_value = d_matrix[row * n + col];
        if (current_value == 0 || current_value == 1) {
            int count[5] = {0};  // Count for each value (0-4)
            
            // Check 3x3 neighborhood with periodic boundary conditions
            for (int i = -1; i <= 1; i++) {
                for (int j = -1; j <= 1; j++) {
                    if (i == 0 && j == 0) continue;  // Skip the center cell
                    int neighbor_row = (row + i + m) % m;
                    int neighbor_col = (col + j + n) % n;
                    int neighbor_value = d_matrix[neighbor_row * n + neighbor_col];
                    count[neighbor_value]++;
                }
            }

            // Find the most frequent value (smallest in case of a tie)
            int max_count = 0;
            int new_value = current_value;
            for (int i = 0; i < 5; i++) {
                if (count[i] > max_count || (count[i] == max_count && i < new_value)) {
                    max_count = count[i];
                    new_value = i;
                }
            }

            // Update the cell if necessary
            if (new_value != current_value) {
                d_new_matrix[row * n + col] = new_value;
                *d_updated = true;
            } else {
                d_new_matrix[row * n + col] = current_value;
            }
        } else {
            d_new_matrix[row * n + col] = current_value;
        }
    }
}

// Host function to update the matrix
void updateMatrix(int* matrix, int m, int n) {
    int* d_matrix;
    int* d_new_matrix;
    bool* d_updated;
    bool updated;

    size_t size = m * n * sizeof(int);

    // Allocate device memory
    hipMalloc((void**)&d_matrix, size);
    hipMalloc((void**)&d_new_matrix, size);
    hipMalloc((void**)&d_updated, sizeof(bool));

    // Copy input matrix to device
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

    do {
        // Reset the updated flag
        updated = false;
        hipMemcpy(d_updated, &updated, sizeof(bool), hipMemcpyHostToDevice);

        // Launch the kernel
        updateMatrixKernel<<<gridDim, blockDim>>>(d_matrix, d_new_matrix, m, n, d_updated);

        // Swap the matrices
        int* temp = d_matrix;
        d_matrix = d_new_matrix;
        d_new_matrix = temp;

        // Check if any updates were made
        hipMemcpy(&updated, d_updated, sizeof(bool), hipMemcpyDeviceToHost);
    } while (updated);

    // Copy the result back to host
    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix);
    hipFree(d_new_matrix);
    hipFree(d_updated);
}

// Main function for testing
int main() {
    int m = 5, n = 5;
    int matrix[25] = {
        0, 1, 2, 3, 4,
        1, 2, 3, 4, 0,
        2, 3, 4, 0, 1,
        3, 4, 0, 1, 2,
        4, 0, 1, 2, 3
    };

    printf("Original matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }

    updateMatrix(matrix, m, n);

    printf("\nUpdated matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }

    return 0;
}